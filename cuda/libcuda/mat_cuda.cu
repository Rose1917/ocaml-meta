#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <caml/bigarray.h>
#include <caml/mlvalues.h>
#include <caml/alloc.h>
#include <sys/time.h>
#include <x86intrin.h>
#include <omp.h>
#include <hip/hip_runtime.h>
#include <>
#include <string.h>
#include "common.h"
#include "boost.h"
#include "mat_cuda.h"

#define BUFFER_SIZE 1000000
//#define DEBUG



double res[BUFFER_SIZE];

__global__ void mul_cuda(double* x, double* y, double* z, int m, int n, int p)
{
    int row = blockIdx.x;
    int col = threadIdx.x;

    double tmp = 0.0;
        for (int i = 0; i < n; i++)
        {
            tmp += x[row * n + i] * y[col + i * p];
        }
        z[row * p + col] = tmp;
}

//matrix operation
CAMLprim value 
cuda_mat_mul (value x,value y){

#ifdef DEBUG
	printf("now in the cuda_mul function\n");
#endif
	
	//obtain the data from value type
	double* x_val =(double*) Caml_ba_data_val(x);
	double* y_val =(double*) Caml_ba_data_val(y);

	//obtain the properties pointer from the value type
	struct caml_ba_array *x_pro =Caml_ba_array_val(x);
	struct caml_ba_array *y_pro =Caml_ba_array_val(y);
	
	//check the shape of the two matrixs
	int x_dimension = x_pro -> num_dims;
	int y_dimension = y_pro -> num_dims;

	if( x_dimension != 2 || y_dimension != 2){
		printf("cuda_mat_mul:dimension error \n");
		printf("left operand dimension number: %d,the right dimension number: %d\n",x_dimension,y_dimension);
		exit(DIMENSION_NOT_QUALIFIED);
	}

#ifdef DEBUG
	printf("first judgement\n");
#endif
	//define the shape of the two matrix
	int x_r,x_c;
	int y_r,y_c;

	//obtain the row and col of x and y
	x_r = x_pro -> dim[0];
	x_c = x_pro -> dim[1];

	y_r = y_pro -> dim[0];
	y_c = y_pro -> dim[1];

	//check if the shape matchs
	if( x_c != y_r ){
		printf("cuda_mat_mul:shape not match\n");
		printf("the shape of left operand %d x %d",x_r,x_c);
		printf("the shape of right operand %d x %d",y_r,y_c);
		exit(SHAPE_NOT_MATCH);
	}	

#ifdef DEBUG
	//traverse the x
	for ( int i = 0; i<x_r ; i++){
		for (int j = 0;j<x_c;j++)
			printf(" %f",x_val[i*x_c+j]);
		printf ("\n");
	}

	//traverse the y
	for ( int i = 0; i<y_r ; i++){
		for (int j = 0;j<y_c;j++)
			printf(" %f",y_val[i*y_c+j]);
		printf ("\n");
	}
#endif

	//so the result matrix shape is x_r * y_c
	long  res_r = x_r;
	long  res_c = y_c;

	//prepare for the result buffer
	if(res_r * res_c > BUFFER_SIZE) {
		printf ("cuda mat_mul:buffer is not enough");
		exit(BUFFER_OVERFLOW);
	}
	
	//init the matrix c
	memset(res,0,sizeof(double) * res_r * res_c);

	//the GPU memory pointer	
	double *x_cuda, *y_cuda, *res_cuda;

	//malloc the memory from gpu
	hipMalloc((void**)&x_cuda, sizeof(double) * x_r * x_c);
	hipMalloc((void**)&y_cuda, sizeof(double) * y_r * y_c);
	hipMalloc((void**)&res_cuda, sizeof(double) * res_r * res_c);

	//set the value
	hipMemcpy(x_cuda, x_val, sizeof(double) * x_r * x_c, hipMemcpyHostToDevice);
	hipMemcpy(y_cuda, y_val, sizeof(double) * y_r * y_c, hipMemcpyHostToDevice);
	hipMemcpy(res_cuda, res, sizeof(double) * res_r * res_c, hipMemcpyHostToDevice);
	
	//we can not know the row and column here
	//so we transfer the two dimensional result matrix to a liner
	dim3 grid (res_c);
	dim3 block(res_r);

	mul_cuda<<<grid, block>>>(x_cuda, y_cuda, res_cuda, x_r, x_c, y_c);

	//copy back the data
	hipMemcpy(res, res_cuda, sizeof(double) * res_r * res_c, hipMemcpyDeviceToHost);

	//all the function and if it works properly, we need to pack it to a bigarray so we can send it back
	return caml_ba_alloc_dims(CAML_BA_FLOAT64|CAML_BA_C_LAYOUT,2,res,res_r,res_c);
}

int main(){
	printf("in mian");
	return 0;
}
